#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "managed_allocator.h"
#include <thrust/generate.h>
#include <thrust/transform.h>
#include <thrust/for_each.h>
#include <algorithm>
#include <random>

template<class T>
using managed_device_vector = thrust::device_vector<T, managed_allocator<T, hipMemAttachGlobal>>;


template<class T>
using managed_host_vector = thrust::device_vector<T, managed_allocator<T, hipMemAttachHost>>;


hipfftComplex gen_rand_cufftComplex() {
   static std::default_random_engine r_engine;
   static std::uniform_real_distribution<float> udist{0.0, 50.0}; // range 0 - 50

   /*hipfftComplex result{(float)udist(r_engine),(float)udist(r_engine)};*/
   return hipfftComplex{(float)udist(r_engine),(float)udist(r_engine)};
}


inline void cout_cufftComplex( const hipfftComplex& val ) {
   std::cout << "{ " << val.x << ", " << val.y << " }\n";
}


int main( int argc, char** argv) {

   int num_vals = 1000;
   int window_size = 40;
   int num_sums = num_vals - window_size;
   
   managed_device_vector<hipfftComplex> vals( num_vals );
   managed_device_vector<hipfftComplex> sums( num_sums );

   for( size_t index = 0; index != vals.size(); ++index ) {
      vals[index] = gen_rand_cufftComplex();
   } 

   std::cout << "Vals:\n"; 
   for( size_t index = 0; index != vals.size(); ++index ) {
      cout_cufftComplex( vals[index] );  
   } 
   std::cout << "\n";

   return 0;
}


